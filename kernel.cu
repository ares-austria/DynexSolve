#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// DYNEXSOLVE
// ----------------------------------------------------------------------------------------------------
// Copyright (c) 2021-2023, The Dynex Project
//
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are
// permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of
//    conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice, this list
//    of conditions and the following disclaimer in the documentation and/or other
//    materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may be
//    used to endorse or promote products derived from this software without specific
//    prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF
// MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL
// THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
// STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF
// THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#pragma comment(lib, "libcurl.lib" )
#pragma comment(lib, "winmm.lib" )
#pragma comment(lib, "ws2_32.lib")
#pragma comment (lib, "zlib.lib")
#pragma comment (lib, "advapi32.lib")
#pragma comment (lib, "crypt32.lib")

#include <memory.h>
#include <stdio.h>
#include <stdlib.h>
#include <signal.h>
#include <math.h>
#include <stdbool.h>
#include <locale.h>
#include "memory.h"
#include <chrono>
#include <map>
#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>

// for cuda:
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <iomanip>

// for curl:
#include <curl/curl.h> 
#include "jsonxx.h"    
CURL* curl; // init curl

// for ftp:
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <errno.h>
#ifdef WIN32
	#include <io.h>
#else
	#include <unistd.h>
#endif

// defines and version:
std::string VERSION = "2.2.5";
std::string REVISION = "b";
//#define GPUDEBUG
#define MAX_NUM_GPUS      	 			32   // maximum supported by nvidia driver
#define HASHRATE_INTERVAL    			20
#define MAX_ATOMIC_ERR       			10
#define MAX_MALLOB_ERR       			5   
#define ADJ_DEFAULT          			1.3
#define MAX_REJECTED_SERIES  			20
#define PRECISION 						double //precision of ODE integration;
#define JOB_TYPE_SAT                    0
#define JOB_TYPE_MILP                   1
#define JOB_TYPE_QUBO                   2
#define JOB_TYPE_MAXSAT                 3
#define JOB_TYPE_FEDERATED_ML           4
#define JOB_TYPE_PRETRAINING_ML         5
#define JOB_TYPE_SUBSET_SUM             6
#define JOB_TYPE_INTEGER_FACTORISATION  7
#define ATOMIC_STATUS_ASSIGNED          0
#define ATOMIC_STATUS_RUNNING           1
#define ATOMIC_STATUS_FINISHED_SOLVED   2
#define ATOMIC_STATUS_FINISHED_UNKNOWN  3
#define ATOMIC_STATUS_INTERRUPTED       4

using namespace std;

#include "log.hpp" 			// logger
#include "picosha3.h" 		// sha3
#include "dynexsolve.hpp" 	// mallob communication handler class * REDACTED *
#include "Dynexchip.cpp" 	// cpu dynex chips class
#include "dynexservice.cpp" // stratum communication handler class * REDACTED *

typedef long long int int64_cu;
typedef unsigned long long int uint64_cu;

enum {
	ASSIGNED = 		0,
	RUNNING = 		1,
	SOLVED = 		2,
	UNKNOWN = 		3,
	CANCELLED = 	4
} ATOMIC_STATUS;

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// global variables:
int n; 							// number of variables
int m; 							// number of clauses
bool* solution; 				// will contain the solution if found (size = n)
int nDevices = 0; 				// number of cuda devices
bool use_multi_gpu = false;		// single/multi GPU usage
std::string JOB_FILENAME = "";  
int num_jobs[MAX_NUM_GPUS] = {0};
int num_jobs_all = 0;

// job data structure:
typedef struct
{
	int threadi;
	int n;
	int m;
	uint32_t xl_max;
	int global;
	int loc;
	int solved; 
	int padding;
	int* cls;
	PRECISION* initial_conditions;
	PRECISION* x;
	PRECISION* dxdt;
	PRECISION t;
	PRECISION stepsize;
	PRECISION dmm_alpha;
	PRECISION dmm_beta;
	PRECISION dmm_gamma;
	PRECISION dmm_delta;
	PRECISION dmm_epsilon;
	PRECISION dmm_zeta;
	PRECISION energy;
} job_struct_2;

typedef struct {
	int loc;
	PRECISION energy;
	uint64_cu steps;
	bool solution[1];
} state_struct;

// vars:
int* cls{};
int* d_cls[MAX_NUM_GPUS]{};
state_struct* d_state[MAX_NUM_GPUS]{};
job_struct_2* d_jobs_2[MAX_NUM_GPUS]{};
job_struct_2* h_jobs_2[MAX_NUM_GPUS]{};
size_t max_heap_size[MAX_NUM_GPUS] = {0};
PRECISION dmm_alpha = 5.0;
PRECISION dmm_beta = 20.0;
PRECISION dmm_gamma = 0.25;
PRECISION dmm_delta = 0.05;
PRECISION dmm_epsilon = 0.1;
PRECISION dmm_zeta = 0.1;
PRECISION init_dt = 0.15;
int CNF_CHECKSUM = 0;
std::string CNF_DOWNLOADURL = "";
std::string CNF_SOLUTIONURL = "";
std::string CNF_SOLUTIONUSER = "";
std::atomic<int> overall_loc{0};
std::atomic<PRECISION> overall_energy{0};
float overall_hashrates[MAX_NUM_GPUS]{0};
int threadsPerBlock[MAX_NUM_GPUS];
int numBlocks[MAX_NUM_GPUS];
std::atomic<bool> atomic_updated{0};
PRECISION factor = 0.0;

// system definitions:
#define MAX_LIT_SYSTEM 3
int max_adj_size = 0;
uint64_cu PARALLEL_RUNS;
bool debug = false;
bool testing = false;
std::string testing_file;
bool DISC_OPERATION = false;
std::vector<int> disabled_gpus;
auto t0 = std::chrono::steady_clock::now();

// mallob definitions:
bool MALLOB_ACTIVE = false;
int JOB_ID = -1; 
std::string MALLOB_NETWORK_ID = "";

// default parameters:
std::string MINING_ADDRESS = ""; 
float rem_hashrate = 0;
int INTENSITY = 0;
float ADJ[MAX_NUM_GPUS] = {0};
bool SKIP = false;
std::string STATS = "";
std::string BUSID = "";
uint8_t network_id[32] = {0};

// stratum
bool stratum = false;
std::string STRATUM_URL = ""; 
int STRATUM_PORT = 0;  
std::string STRATUM_PAYMENT_ID = ""; 
std::string STRATUM_PASSWORD  = ""; 
int  STRATUM_DIFF = 0;

// Dynex classes:
std::atomic_bool dynex_quit_flag;
Dynex::dynexchip dynexchip;
Dynexservice::dynexservice dynexservice;

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// system helper functions
///////////////////////////////////////////////////////////////////////////////////////////////////////////

std::string log_time() {
	auto t = std::time(nullptr);
	auto tm = *std::localtime(&t);
	std::ostringstream oss;
	oss << std::put_time(&tm, "%d-%m-%Y %H:%M:%S");
	auto str = oss.str();
	return str;
}

// file existing function
inline bool file_exists(const std::string& name) {
	if (FILE *file = fopen(name.c_str(), "r")) {
		fclose(file);
		return true;
	} else {
		return false;
	}
}

std::vector<std::string> split(std::string text, char delim) {
	std::string line;
	std::vector<std::string> vec;
	std::stringstream ss(text);
	while(std::getline(ss, line, delim)) {
		vec.push_back(line);
	}
	return vec;
}

template <typename T>
T atomic_fetch_min(atomic<T>* pv, typename atomic<T>::value_type v) noexcept {
	auto t = pv->load(std::memory_order_relaxed);
	while (std::min(v, t) != t) {
		if (pv->compare_exchange_weak(t, v, std::memory_order_relaxed, std::memory_order_relaxed))
			break;
	}
	return t;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
/// FTP FUNCTIONS
///////////////////////////////////////////////////////////////////////////////////////////////////////////
static size_t read_callback(char *ptr, size_t size, size_t nmemb, void *stream)
{
	std::size_t retcode = std::fread( ptr, size, nmemb, static_cast<std::FILE*>(stream) );
	return retcode;
}

struct FtpFile {
	const char *filename;
	FILE *stream;
};

static int my_fwrite(void *buffer, size_t size, size_t nmemb, void *stream)
{
	struct FtpFile *out = (struct FtpFile *)stream;
	if(!out->stream) {
		out->stream = fopen(out->filename, "wb");
		if(!out->stream)
			return 0;
	}
	return fwrite(buffer, size, nmemb, out->stream);
}

bool upload_file(const std::string filename) {
	CURLcode res;
	FILE *hd_src;
	struct stat file_info;
	unsigned long fsize;
	bool ret = false;

	/* get the file size of the local file */
	if(stat(filename.c_str(), &file_info)) {
		LogTS(TEXT_BRED) << "[ERROR] Couldn't open '" << filename.c_str() << "': " <<  strerror(errno) << std::endl;
		return false;
	}
	fsize = (unsigned long)file_info.st_size;

	LogTS() << "[INFO] Local file size: " << fsize << " bytes" << std::endl;

	/* get a FILE * of the same file */
	hd_src = fopen(filename.c_str(), "rb");

	/* In windows, this will init the winsock stuff */
	//curl_global_init(CURL_GLOBAL_ALL);

	/* get a curl handle */
	curl = curl_easy_init();
	if(curl) {
		/* we want to use our own read function */
		curl_easy_setopt(curl, CURLOPT_READFUNCTION, read_callback);

		/* enable uploading */
		curl_easy_setopt(curl, CURLOPT_UPLOAD, 1L);

		/* specify target */
		std::string remoteurl = CNF_SOLUTIONURL + filename;
		curl_easy_setopt(curl, CURLOPT_URL, remoteurl.c_str());
		curl_easy_setopt(curl, CURLOPT_USERPWD, CNF_SOLUTIONUSER.c_str());
		curl_easy_setopt(curl, CURLOPT_SSL_VERIFYPEER, 0L);
		curl_easy_setopt(curl, CURLOPT_SSL_VERIFYHOST, 0L);
		curl_easy_setopt(curl, CURLOPT_IPRESOLVE, CURL_IPRESOLVE_V4);
		//curl_easy_setopt(curl, CURLOPT_VERBOSE, 1L);

		/* now specify which file to upload */
		curl_easy_setopt(curl, CURLOPT_READDATA, hd_src);
		curl_easy_setopt(curl, CURLOPT_INFILESIZE_LARGE, (curl_off_t)fsize);

		/* Now run off and do what you have been told! */
		res = curl_easy_perform(curl);

		/* Check for errors */
		if(res != CURLE_OK) {
			LogTS(TEXT_BRED) << "[ERROR] UPLOAD FAILED: " << curl_easy_strerror(res) << std::endl;
		} else {
			ret = true;
		}
		/* always cleanup */
		curl_easy_cleanup(curl);
	}
	fclose(hd_src); /* close the local file */
	return ret;
}

bool download_file(const std::string filename) {
	CURLcode res;

	struct FtpFile ftpfile = {
		filename.c_str(),
		NULL
	};

	curl = curl_easy_init();
	if (curl) {
		std::string remoteurl = "https://jobs.dynexcoin.org/" + filename; 
		if (debug) LogTS() << "[INFO] Downloading " << remoteurl << "..." << std::endl;
		curl_easy_setopt(curl, CURLOPT_URL, remoteurl.c_str());
		curl_easy_setopt(curl, CURLOPT_IPRESOLVE, CURL_IPRESOLVE_V4);
		curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, my_fwrite);
		curl_easy_setopt(curl, CURLOPT_WRITEDATA, &ftpfile);
		curl_easy_setopt(curl, CURLOPT_FOLLOWLOCATION, 1L);
		res = curl_easy_perform(curl);

		long status_code = 0;
		curl_easy_getinfo(curl, CURLINFO_RESPONSE_CODE, &status_code);
		curl_easy_cleanup(curl);

		if (ftpfile.stream) {
			fclose(ftpfile.stream);
			if (res == CURLE_OK && status_code == 200) return true;
			std::remove(filename.c_str()); // delete broken file
		}
		if (CURLE_OK != res) {
			LogTS(TEXT_BRED) << "[ERROR] " << curl_easy_strerror(res) << std::endl;
			return false;
		}
		if (status_code != 200) {
			LogTS(TEXT_BRED) << "[ERROR] HTTP STATUS CODE: " << status_code << std::endl;
			return false;
		}
	}
	LogTS(TEXT_BRED) << "[ERROR] CURL FAILED" << std::endl;
	return false;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess) {
		fprintf(stdout, " GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
bool load_cnf(const std::string& filename, const std::string& filehash = {}) {
	n = 0;
	m = 0;
	int checksum = 0;
	std::string line;
	auto sha3_256 = picosha3::get_sha3_generator<256>();
	auto t1 = std::chrono::steady_clock::now();

	std::ifstream file(filename);
	if (!file.is_open()) {
		LogTS(TEXT_BRED) << "[ERROR] FILE NOT FOUND: " << filename << std::endl;
		return false;
	}

	LogTS() << "[INFO] LOADING FILE: " << filename << std::endl;

	while (std::getline(file, line)) {
		sha3_256.process(line.cbegin(), line.cend());
		if (std::sscanf(line.c_str(), "p cnf %u %u", &n, &m) == 2) {
			break;
		}
	}

	if (!n || !m) {
		LogTS(TEXT_BRED) << "[ERROR] INVALID FORMAT" << std::endl;
		return false;
	}

	LogTS() << "[INFO] VARIABLES : " << n << std::endl;
	LogTS() << "[INFO] CLAUSES   : " << m << std::endl;
	LogTS() << "[INFO] RATIO     : " << ((double)m / n) << std::endl;

	/// reserve  memory:
	cls = (int*)calloc((size_t)m * MAX_LIT_SYSTEM, sizeof(int));

	// read CNF:
	int res[MAX_LIT_SYSTEM+1];
	int lit;
	int i = -1;
	while (std::getline(file, line)) {
		sha3_256.process(line.cbegin(), line.cend());

		lit = std::sscanf(line.c_str(), "%d %d %d %d", &res[0], &res[1], &res[2], &res[3]); // MAX_LIT_SYSTEM + 1
		if (lit == 0) continue; // skip comments and empty lines
		i++;
		if (i == m) break;

		// check max amount
		if (lit > MAX_LIT_SYSTEM && res[MAX_LIT_SYSTEM] != 0) {
			LogRTS(TEXT_BRED) << "[ERROR] CLAUSE " << i << " HAS " << lit << " LITERALS (" << MAX_LIT_SYSTEM << " ALLOWED)" << std::endl;
			return false;
		}

		for (int j = lit; j > 0; j--) {
			if (res[j-1] == 0) {
				lit--;
			} else if (res[j-1] > n || res[j-1] < -n) {
				LogRTS(TEXT_BRED) << "[INFO] CLAUSE " << i << " HAS BAD LITERAL " << res[j-1] << " (" << n << " ALLOWED)" << std::endl;
				return false;
			}
		}

		// do not allow zero
		if (lit == 0 || res[0] == 0) {
			LogRTS() << "[INFO] CLAUSE: " << i << " HAS NO LITERALS" << std::endl;
			return false;
		}

		if (debug && i % 100000 == 0) {
			LogRTS() << "[INFO] LOADING   : " << int(100.0 * (i + 1) / m) << "% " << std::flush;
		}

		for (int j = 0; j < MAX_LIT_SYSTEM; j++) {
			if (j >= lit) res[j] = res[j-1];
			cls[i * MAX_LIT_SYSTEM + j] = res[j];
			checksum += res[j];
		}
	}

	file.close();

	sha3_256.finish();
	std::array<uint8_t, picosha3::bits_to_bytes(256)> hash{};
	sha3_256.get_hash_bytes(hash.begin(), hash.end());
	std::string sha3hash = picosha3::bytes_to_hex_string(hash); // cat job.cnf | tr -d '\n' | rhash --sha3-256 -
	
	if (debug) {
		LogRTS() << "[INFO] LOADING   : " << int(100.0 * (i + 1) / m) << "% " << std::endl;
	}

	if (i + 1 != m) {
		LogRTS(TEXT_BRED) << "[ERROR] UNEXPECTED END OF FILE: " << i << " (" << m << " EXPECTED)" << std::endl;
		return false;
	}

	auto t2 = std::chrono::steady_clock::now();
	float dur = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
	if (debug) LogTS() << "[INFO] LOADED IN " << dur << "ms" << std::endl;

	if (debug) {
		LogTS(TEXT_GRAY) << "[INFO] FIRST 10 CLAUSES:" << std::endl;
		for (i = 0; i < 10; i++) {
			LogTS(TEXT_GRAY) << "[INFO] CLAUSE " << i << ": ";
			for (int j = 0; j < MAX_LIT_SYSTEM; j++) {
				Log(TEXT_GRAY) << cls[i * MAX_LIT_SYSTEM + j] << " ";
			}
			Log() << std::endl;
		}
	}

	// verification with checksum:
	if (!testing) {
		if (filehash != sha3hash) {
			LogTS(TEXT_BRED) << "[ERROR] INCORRECT PROBLEM FILE" << std::endl;
			return false;
		}
	} else {
		checksum = checksum * m / n;
		if (debug) LogTS(TEXT_BCYAN) << "[INFO] " << sha3hash << " | " << checksum << std::endl;
	}

	return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Query devices
///////////////////////////////////////////////////////////////////////////////////////////////////////////
int query_devices(int device_id) {
	int nDevices;
	if (hipGetDeviceCount(&nDevices) != hipSuccess) {
		LogTS(TEXT_BRED) << "[ERROR] Unable to get GPU amount" << std::endl;
		return 0;
	}
	int runtimeVersion;
	hipRuntimeGetVersion(&runtimeVersion);
	int driverVersion;
	hipDriverGetVersion(&driverVersion);
	LogTS() << "[INFO] CUDA RUNTIME: " << runtimeVersion/1000 << "." << runtimeVersion%1000/10 << std::endl;
	LogTS() << "[INFO] CUDA DRIVER:  " << driverVersion/1000 << "." << driverVersion%1000/10 << std::endl;
	LogTS() << "[INFO] FOUND " << nDevices << " INSTALLED GPU(s)" << std::endl;

	if (device_id >= 0 && device_id < nDevices) LogTS(TEXT_SILVER) << "[INFO] USING GPU DEVICE " << device_id << std::endl;

	BUSID = "";
	std::string adj_str;
	for (int i = (device_id==-1?0:device_id); i < nDevices; i++) {
		if (std::find(disabled_gpus.begin(), disabled_gpus.end(), i) != disabled_gpus.end()) continue; // skip disabled
		hipSetDevice(i);
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		adj_str = "";
		if (ADJ[i]) {
			adj_str = " ADJ = ";
			adj_str.append(std::to_string(ADJ[i]));
		}
		LogTS(TEXT_BCYAN) << "[GPU " << i << "] " << std::setfill('0') << std::setw(2) << std::hex << devProp.pciBusID << ":"
			<< std::setw(2) << devProp.pciDeviceID << " " << devProp.name << " " << std::dec << devProp.totalGlobalMem/1024/1024 << " MB ("
			<< devProp.major << "." << devProp.minor << ")" << adj_str.c_str() << std::endl;

		BUSID.append(BUSID == "" ? "[" : ",").append(std::to_string(devProp.pciBusID));
		if (device_id != -1) break;
	}
	if (BUSID != "") BUSID.append("]");
	return nDevices;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
/// functions
///////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void gpu_set_initial_conditions(const int dev, const int n, const int m, const int numchips, job_struct_2* d_jobs, state_struct* d_state,
	const PRECISION stepsize, const PRECISION dmm_alpha, const PRECISION dmm_beta, const PRECISION dmm_gamma, const PRECISION dmm_delta,
	const PRECISION dmm_epsilon, const PRECISION dmm_zeta) {

	for (int threadi = blockIdx.x * blockDim.x + threadIdx.x; threadi < numchips; threadi += blockDim.x * gridDim.x) {
		// globals
		if (threadi == 0) {
			d_state[0].steps = 0;
			d_state[0].energy = (PRECISION)m;
			d_state[0].loc = m;
			for (int i = 0; i < n; i++) {
				d_state[0].solution[i] = 0;
			}
		} else {
			for (int i = 0; i < m*3; i++) {
				d_jobs[threadi].cls[i] = d_jobs[0].cls[i]; // copy from thread 0
			}
		}

		for (int i = 0; i < n + m*2; i++) {
			d_jobs[threadi].dxdt[i] = 0.0;
			d_jobs[threadi].x[i] = 0.0;
		}

		// init job:
		d_jobs[threadi].t = 0.0;
		d_jobs[threadi].threadi = threadi;
		d_jobs[threadi].n = n;
		d_jobs[threadi].m = m;
		d_jobs[threadi].xl_max = 10000 * m;
		d_jobs[threadi].global = m;
		d_jobs[threadi].loc = m;
		d_jobs[threadi].energy = (PRECISION)m;
		d_jobs[threadi].solved = false;
		d_jobs[threadi].stepsize = stepsize; 
		d_jobs[threadi].dmm_alpha = dmm_alpha * (0.5 + (numchips - threadi + 1)/(PRECISION)numchips) ; //alpha distribution
		d_jobs[threadi].dmm_beta = dmm_beta;
		d_jobs[threadi].dmm_gamma = dmm_gamma;
		d_jobs[threadi].dmm_delta = dmm_delta;
		d_jobs[threadi].dmm_epsilon = dmm_epsilon;
		d_jobs[threadi].dmm_zeta = dmm_zeta;
	}
	
}
//------------------------------------------------------------------------------------------------------------------------------

__global__
void gpu_load_initial_conditions(const int dev, const int numchips, job_struct_2* d_jobs, bool use_random, bool testing) {
	for (int threadi = blockIdx.x * blockDim.x + threadIdx.x; threadi < numchips; threadi += blockDim.x * gridDim.x) {
		if (use_random) {
			// assign random initial states for Voltages:
			hiprandState state;
			hiprand_init((unsigned long long)clock(), threadi, 0, &state);
			for (int i = 0; i < d_jobs[threadi].n; i++) {
				float RANDOM = hiprand_uniform( &state );

				// three states:
				d_jobs[threadi].initial_conditions[i] = 0.0;
				if (RANDOM <= 0.33) d_jobs[threadi].initial_conditions[i] = -1.0;
				if (RANDOM >= 0.66) d_jobs[threadi].initial_conditions[i] = +1.0;
				
				// default 3 lanes:
				if (threadi == 0) d_jobs[threadi].initial_conditions[i] = 0.0;
				if (threadi == 1) d_jobs[threadi].initial_conditions[i] = 1.0;
				if (threadi == 2) d_jobs[threadi].initial_conditions[i] = -1.0;
				
			}
			// set Xs:
			for (int i=d_jobs[threadi].n; i<d_jobs[threadi].n + d_jobs[threadi].m; i++) {
				d_jobs[threadi].x[i] = 0.0;
			}
			// set Xl:
			for (int i=d_jobs[threadi].n+d_jobs[threadi].m; i<d_jobs[threadi].n + d_jobs[threadi].m*2; i++) {
				d_jobs[threadi].x[i] = 1.0;
			}
		}
		for (int i=0; i<d_jobs[threadi].n; i++) {
			d_jobs[threadi].x[i] = d_jobs[threadi].initial_conditions[i];
		}
	}
}

//------------------------------------------------------------------------------------------------------------------------------
__global__
void gpu_reset_dxdt(const int dev, const int numchips, job_struct_2* d_jobs) {
	for (int threadi = blockIdx.x * blockDim.x + threadIdx.x; threadi < numchips; threadi += blockDim.x * gridDim.x) {
		job_struct_2* job = &d_jobs[threadi];
		// reset dxdt:
		for (int i = 0; i < job->n + job->m*2; i++) {
			job->dxdt[i] = 0.0;
		}
		// reset loc:
		job->loc = job->m;
		// reset energy:
		job->energy = 0.0;
	}
}

//------------------------------------------------------------------------------------------------------------------------------
__global__
void gpu_step(const int dev, const int numchips, job_struct_2* d_jobs, state_struct* d_state) {
	for (int threadi = blockIdx.x * blockDim.x + threadIdx.x; threadi < numchips; threadi += blockDim.x * gridDim.x) {
		job_struct_2* job = &d_jobs[threadi];
		const PRECISION dmm_alpha_f 	= job->dmm_alpha;
		const PRECISION dmm_beta_f 		= job->dmm_beta;
		const PRECISION dmm_gamma_f 	= job->dmm_gamma;
		const PRECISION dmm_delta_f 	= job->dmm_delta;
		const PRECISION dmm_epsilon_f 	= job->dmm_epsilon;
		const PRECISION dmm_zeta_f 		= job->dmm_zeta;
		const uint32_t xl_max_f 		= job->xl_max;
		const int n_f 					= job->n;
		const int m_f 					= job->m;
		// loop through each clause:
		for (int clause = 0; clause < m_f; clause++) {
			const int a = job->cls[clause*MAX_LIT_SYSTEM+0];
			const int b = job->cls[clause*MAX_LIT_SYSTEM+1];
			const int c = job->cls[clause*MAX_LIT_SYSTEM+2];
			const int liti = abs(a);
			const int litj = abs(b);
			const int litk = abs(c);
			const PRECISION Qi = (a > 0)? 1.0:-1.0; // +1 if literal is >0, otherwise -1
			const PRECISION Qj = (b > 0)? 1.0:-1.0; // +1 if literal is >0, otherwise -1
			const PRECISION Qk = (c > 0)? 1.0:-1.0; // +1 if literal is >0, otherwise -1
			PRECISION Xs = job->x[clause+n_f]; if (Xs<0.0) Xs = 0.0; else if (Xs>1.0) Xs = 1.0; //Xs bounds
			PRECISION Xl = job->x[clause+n_f+m_f]; if (Xl<1.0) Xl = 1.0; else if (Xl>xl_max_f) Xl = PRECISION(xl_max_f); //Xl bounds
			// 3-sat:
			PRECISION Vi = job->x[liti-1]; if (Vi<-1.0) Vi = -1.0; else if (Vi>1.0) Vi = 1.0; //V bounds
			PRECISION Vj = job->x[litj-1]; if (Vj<-1.0) Vj = -1.0; else if (Vj>1.0) Vj = 1.0; //V bounds
			PRECISION Vk = job->x[litk-1]; if (Vk<-1.0) Vk = -1.0; else if (Vk>1.0) Vk = 1.0; //V bounds
			const PRECISION i = 1.0 - Qi*Vi;
			const PRECISION j = 1.0 - Qj*Vj;
			const PRECISION k = 1.0 - Qk*Vk;
			PRECISION C = fmin(i, fmin(j, k)) / 2.0; if (C < 0.0) C = 0.0; else if (C > 1.0) C = 1.0;
			//voltages:
			const PRECISION Gi = Qi * fmin(j, k) / 2.0;
			const PRECISION Gj = Qj * fmin(i, k) / 2.0;
			const PRECISION Gk = Qk * fmin(i, j) / 2.0;
			PRECISION Ri, Rj, Rk;
			if (C != i/2.0 ) {Ri = 0.0;} else {Ri = (Qi - Vi) / 2.0;}
			if (C != j/2.0 ) {Rj = 0.0;} else {Rj = (Qj - Vj) / 2.0;}
			if (C != k/2.0 ) {Rk = 0.0;} else {Rk = (Qk - Vk) / 2.0;}
			const PRECISION tmp1 = Xl * Xs;
			const PRECISION tmp2 = (1.0 + dmm_zeta_f * Xl) * (1.0 - Xs);
			job->dxdt[liti-1] += tmp1*Gi + tmp2 * Ri;
			job->dxdt[litj-1] += tmp1*Gj + tmp2 * Rj;
			job->dxdt[litk-1] += tmp1*Gk + tmp2 * Rk;

			// clause satsfied?
			if (C < 0.5) job->loc--;
			// update energy:
			job->energy += C;
			// Calculate Xs:
			job->dxdt[n_f + clause] = dmm_beta_f * (Xs + dmm_epsilon_f) * (C - dmm_gamma_f);
			// Calculate Xl:
			job->dxdt[n_f + m_f + clause] = dmm_alpha_f * (C - dmm_delta_f);
		}

		// only for debugging --------------------------------------------------------------------------------------------
		#ifdef GPUDEBUG
			if (job->loc < d_state[0].loc || job->energy < d_state[0].energy) {
			printf("CHIP %d: T=%f loc=%d energy=%f (global=%d) solved=%d alpha=%.5f stepsize=%.5f\n",threadi, job->t,  job->loc, job->energy, job->global, job->solved, job->dmm_alpha, job->stepsize);
			}
		#endif
		
		// ---------------------------------------------------------------------------------------------------------------

		// better global?
		if (job->loc < job->global) {
			job->global = job->loc;
			atomicMin(&d_state[0].loc, job->loc);
		}

		// globals?
		if (job->energy < d_state[0].energy) {
			d_state[0].energy = job->energy;
		}

		// solution found?
		if (job->loc == 0) {
			job->solved = true;
			//move to d_solution
			for (int i=0; i < job->n; i++) {
				d_state[0].solution[i] = (job->x[i] >= 0) ? true : false;
			}
		}

		// update steps:
		if (threadi == 0) d_state[0].steps++;

		// update time:
		job->t += job->stepsize;
	}
}

//------------------------------------------------------------------------------------------------------------------------------
__global__
void gpu_euler(const int dev, const int numchips, job_struct_2* d_jobs) {
	for (int threadi = blockIdx.x * blockDim.x + threadIdx.x; threadi < numchips; threadi += blockDim.x * gridDim.x) {
		job_struct_2* job = &d_jobs[threadi];
		const int n_f = job->n;
		const int m_f = job->m;
		const PRECISION stepsize_f = job->stepsize;
		const uint32_t xl_max_f = job->xl_max;

		for (int i = 0; i < n_f; i++) {
			// euler step:
			job->x[i] += stepsize_f * job->dxdt[i];
			// bounded variables:
			if (job->x[i] < -1.0) job->x[i] = -1.0; else if (job->x[i] > 1.0) job->x[i] =  1.0;
		}

		for (int i = n_f; i < n_f + m_f; i++) {
			// euler step:
			job->x[i] += stepsize_f * job->dxdt[i];
			// bounded variables:
			if (job->x[i] < 0.0) job->x[i] = 0.0; else if (job->x[i] > 1.0) job->x[i] = 1.0;
		}

		for (int i = n_f + m_f; i < n_f + m_f*2; i++) {
			// euler step:
			job->x[i] += stepsize_f * job->dxdt[i];
			// bounded variables:
			if (job->x[i] < 1.0) job->x[i] = 1.0; else if (job->x[i] > xl_max_f) job->x[i] = PRECISION(xl_max_f);
		}
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// new work, initiate state and load to GPU ---------------------------------------------------------------
int init_states_2(int device_id, int maximum_jobs, int use_random) {

	LogTS(TEXT_BCYAN) << "[INFO] INITIALIZING GPU STATES..." << std::endl;

	// base:
	uint64_cu mem_req = 0;
	mem_req += m * sizeof(int) * MAX_LIT_SYSTEM; //d_cls
	mem_req += sizeof(state_struct); //d_state
	mem_req += n * sizeof(bool); //d_solution
	mem_req += 1024; // reserved

	// per job:
	uint64_cu mem_job = 0;
	mem_job += sizeof(job_struct_2);
	mem_job += n * sizeof(PRECISION); //initial_conditions
	mem_job += (n + 2*m) * sizeof(PRECISION); //x
	mem_job += (n + 2*m) * sizeof(PRECISION); //dxdt
	mem_job += (3*m) * sizeof(int); //cls

	LogTS() << "[INFO] BASE MEMORY REQUIRED: " << mem_req << " BYTES" << std::endl;
	LogTS() << "[INFO] MIN MEMORY REQUIRED PER DYNEX CHIP: " << mem_job << " BYTES" << std::endl;

	LogTS() << "[INFO] SETTING MAX HEAP SIZES FOR GPUs..." << std::endl;
	// fitting jobs:
	int jobs_possible_all = 0;
	for (int dev = 0; dev < nDevices; dev++) {
		max_heap_size[dev] = 0;
		if (use_multi_gpu) device_id = dev;
		// only not disabled gpus:
		if (std::find(disabled_gpus.begin(), disabled_gpus.end(), device_id) != disabled_gpus.end()) continue;
		gpuErrchk(hipSetDevice(device_id));
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

		uint64_cu mem_job_gpu = abs(mem_job*ADJ[dev]);
		size_t free, total; //tmp vars
		hipMemGetInfo(&free, &total);
		size_t malloc_limit = free;
		hipDeviceSetLimit(hipLimitMallocHeapSize, malloc_limit);
		hipDeviceGetLimit(&max_heap_size[dev], hipLimitMallocHeapSize);
		LogTS() << "[GPU " << device_id << "] MAX HEAP: " << max_heap_size[dev] << " BYTES" << std::endl;
		if (max_heap_size[dev] < free) max_heap_size[dev] = free;
		if (max_heap_size[dev] <= mem_req + 2*mem_job_gpu) {
			LogTS() << "[GPU " << device_id << "] LOW MEMORY AVAILABLE - DISABLE" << std::endl;
			disabled_gpus.push_back(dev);
		} else {
			jobs_possible_all += (int)((max_heap_size[dev] - mem_req)/mem_job_gpu);
		}
	}

	LogTS() << "[INFO] MAX DYNEX CHIPS FITTING IN MEMORY (ALL GPUs): " << jobs_possible_all << std::endl;

	// num_jobs_all -> total jobs over all gpus:
	int num_jobs_all = jobs_possible_all;
	if (num_jobs_all > maximum_jobs) num_jobs_all = maximum_jobs; // user defined max #jobs

	if (!testing) {
		std::vector<std::string> p3;
		p3.push_back(MALLOB_NETWORK_ID);
		p3.push_back(std::to_string(JOB_ID));
		p3.push_back(std::to_string(num_jobs_all));
		p3.push_back(VERSION);
		jsonxx::Object o1 = mallob_mpi_command("cap", p3, 60);
		if (o1.get<jsonxx::Boolean>("updated")) {
			LogTS(TEXT_BGREEN) << "[MALLOB] UPDATING CAPACITY: SUCCESS" << std::endl;
		} else {
			LogTS(TEXT_BRED) << "[MALLOB] UPDATING CAPACITY: FAILED " << std::endl;
			return 0;
		}
	}

	LogTS(TEXT_BCYAN) << "[INFO] PREPARING " << num_jobs_all << " DYNEX CHIPS..." << std::endl;

	// loop through all GPUs:
	int num_jobs_free = num_jobs_all;

	for (int dev = 0; dev < nDevices; dev++) {
		if (use_multi_gpu) device_id = dev;
		// only not disabled gpus:
		if (std::find(disabled_gpus.begin(), disabled_gpus.end(), device_id) != disabled_gpus.end()) continue;

		gpuErrchk(hipSetDevice(device_id));
		// calculate number of jobs to be created => num_jobs[dev]:
		int jobs_possible = 0;
		jobs_possible = (int)((max_heap_size[dev] - mem_req)/abs(mem_job*ADJ[dev]));
		// less jobs than space here?
		if (jobs_possible > num_jobs_free) {
			jobs_possible = num_jobs_free;
			nDevices = dev + 1; // reduce number of devices needed
		}
		num_jobs[dev] = jobs_possible;
		num_jobs_free = num_jobs_free - num_jobs[dev];

		LogTS() << "[GPU " << device_id << "] PREPARING " << num_jobs[dev] << " DYNEX CHIPS..." << std::endl;

		/// INIT MEMORY WITH KERNEL: ------------------------------------------------------------------------------------------
		LogTS() << "[GPU " << device_id << "] ALLOCATING MEMORY... " << std::endl;
		// create h_jobs and copy to d_jobs:
		int jobs_bytes = num_jobs[dev] * sizeof(job_struct_2);
		h_jobs_2[dev] = (job_struct_2*)calloc(num_jobs[dev], sizeof(job_struct_2));
		//copy jobs over to GPU (including sub arrays):
		uint64_cu mem_reserved = 0;
		for (int i = 0; i < num_jobs[dev]; i++) {
			gpuErrchk(hipMalloc(&(h_jobs_2[dev][i].initial_conditions), (n) * sizeof(PRECISION)));
			gpuErrchk(hipMalloc(&(h_jobs_2[dev][i].x), (n + 2*m) * sizeof(PRECISION)));
			gpuErrchk(hipMalloc(&(h_jobs_2[dev][i].dxdt), (n + 2*m) * sizeof(PRECISION)));
			gpuErrchk(hipMalloc(&(h_jobs_2[dev][i].cls), (3*m) * sizeof(int)));
			mem_reserved += (n + 2*m) * sizeof(PRECISION) * 2 + n * sizeof(PRECISION);
		}

		gpuErrchk(hipMalloc((void**)&d_state[dev], sizeof(state_struct) + n*sizeof(bool)));
		gpuErrchk(hipMalloc((void**)&d_jobs_2[dev], jobs_bytes)); //reserve memory for all jobs

		// cls, d_jobs:
		LogTS() << "[GPU " << device_id << "] COPYING PROBLEM... " << std::endl;
		gpuErrchk(hipMemcpy(d_jobs_2[dev], h_jobs_2[dev], jobs_bytes, hipMemcpyHostToDevice));
		// copy to thread 0
		gpuErrchk(hipMemcpy(h_jobs_2[dev][0].cls, cls, m * MAX_LIT_SYSTEM * sizeof(int), hipMemcpyHostToDevice));

		free(h_jobs_2[dev]);
		
		size_t free, total;
		hipMemGetInfo(&free, &total);
	}

	// ---------------------------------------------------------------------------------------------------------------------
	// set initial conditions:
	LogTS() << "[INFO] SETTING INITIAL CONDITIONS FOR ODE INTEGRATION AT T=0..." << std::endl;

	for (int dev = 0; dev < nDevices; dev++) {
		if (use_multi_gpu) device_id = dev;
		// only not disabled gpus:
		if (std::find(disabled_gpus.begin(), disabled_gpus.end(), device_id) == disabled_gpus.end()) {
			gpuErrchk(hipSetDevice(device_id));
			gpu_set_initial_conditions <<< numBlocks[dev], threadsPerBlock[dev] >>> (dev, n, m, num_jobs[dev], d_jobs_2[dev], d_state[dev], init_dt, dmm_alpha, dmm_beta, dmm_gamma, dmm_delta, dmm_epsilon, dmm_zeta);
		}
	}

	for (int dev = 0; dev < nDevices; dev++) {
		if (use_multi_gpu) device_id = dev;
		// only not disabled gpus:
		if (std::find(disabled_gpus.begin(), disabled_gpus.end(), device_id) == disabled_gpus.end()) {
			gpuErrchk(hipSetDevice(device_id));
			gpuErrchk(hipDeviceSynchronize()); // wait for previous
			gpu_load_initial_conditions <<< numBlocks[dev], threadsPerBlock[dev] >>> (dev, num_jobs[dev], d_jobs_2[dev], use_random, testing);
		}
	}

	for (int dev = 0; dev < nDevices; dev++) {
		if (use_multi_gpu) device_id = dev;
		if (std::find(disabled_gpus.begin(), disabled_gpus.end(), dev) == disabled_gpus.end()) {
			gpuErrchk(hipSetDevice(device_id));
			gpuErrchk(hipDeviceSynchronize());
			// check init was really done
			state_struct h_state[1] = {0};
			gpuErrchk(hipMemcpy(h_state, d_state[dev], sizeof(state_struct), hipMemcpyDeviceToHost));
			if (h_state[0].loc == m && h_state[0].energy == (PRECISION)m) {
				LogTS(TEXT_SILVER) << "[GPU " << device_id << "] INITIALIZED" << std::endl;
			} else {
				LogTS(TEXT_BRED) << "[GPU " << device_id << "] INITIALIZATION FAILED" << std::endl;
				//return 0;
				num_jobs_all = 0; // exit later
			}
		}
	}
	return num_jobs_all;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
void gpu_reporting_runner(int device_id, int num_jobs_all) {
	auto t1 = std::chrono::steady_clock::now();
	uint32_t rem_unixts = 0, errors = 0;
	uint32_t invalid_timestamp_cnt = 0;

	std::this_thread::sleep_for(std::chrono::seconds(HASHRATE_INTERVAL));

	while (!dynex_quit_flag) {
		// screen output:
		std::string gpustats = "";
		int overall_loc_2 = overall_loc;
		int overall_energy_2 = overall_energy;
		float total_hashrate = 0.0;

		for (int dev = 0; dev < nDevices; dev++) {
			if (use_multi_gpu) device_id = dev;
			// only not disabled gpus:
			if (std::find(disabled_gpus.begin(), disabled_gpus.end(), device_id) == disabled_gpus.end()) {
				total_hashrate += overall_hashrates[dev];
				gpustats.append(gpustats == "" ? "[" : ",").append(std::to_string(overall_hashrates[dev]));
			}
		}
		if (gpustats != "") gpustats.append("]");

		auto t2 = std::chrono::steady_clock::now();
		float uptime = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count()/1000.0;

		int hashrate_disp = (int)total_hashrate;
		LogTS(TEXT_SILVER) << "[GPU *] DYNEX CHIPS " << num_jobs_all << " | LOCAL MINIMA " << overall_loc_2 <<
			" | LOWEST ENERGY " << std::fixed << std::setprecision(2) << overall_energy_2 <<
			" | TOTAL HASHRATE " << hashrate_disp << " | UPTIME " << int(uptime) << std::endl;

		if (stratum) {
			uint64_t hashes = 0;
			uint32_t acc = 0, rej = 0, sta = 0, rej_series = 0;;
			dynexservice.getstats(&hashes, &acc, &rej, &sta, &rej_series);
			if (rej_series > MAX_REJECTED_SERIES) {
				LogTS(TEXT_BRED) << "[ERROR] MORE THAN " << MAX_REJECTED_SERIES << " REJECTED SHARES IN A ROW. QUITTING." << std::endl;
				dynex_quit_flag = true;
			}

			float hr = (uptime && hashes > 500) ? (hashes / uptime) : 0;
			LogTS(TEXT_BMAGENTA) << "[INFO] POOL HASHRATE " << int(hr) << " | ACCEPTED " << acc << " | REJECTED " << rej
				<< " | STALE " << sta << " | UPTIME " << int(uptime) << std::endl;
			if (STATS != "") {
				std::ofstream fout(STATS.c_str());
				fout << "{ \"ver\": \"" << VERSION << REVISION << "\", \"avg\": " << int(hr) << ", \"hr\": " << int(total_hashrate)
					<< ", \"ac\": " << acc << ", \"rj\": " << rej << ", \"st\": " << sta << ",\"gpu\": " << (gpustats==""?"null":gpustats)
					<< ", \"bus_numbers\": " << (BUSID==""?"null":BUSID) << ", \"uptime\": " << int(uptime) << " } " << std::endl;
				fout.close();
			}
		}

		if (!testing) {
			// atomic update:
			std::vector<std::string> p1;
			p1.push_back(MALLOB_NETWORK_ID);
			p1.push_back(std::to_string(1)); //status = running
			p1.push_back(std::to_string(overall_loc_2));
			p1.push_back(std::to_string(overall_energy_2));
			p1.push_back(std::to_string(total_hashrate));
			jsonxx::Object o1 = mallob_mpi_command("ato", p1, 60);
			if (o1.get<jsonxx::Boolean>("updated")) {
				errors = 0;
				LogTS() << "[MALLOB] ATOMIC STATUS UPDATED" << std::endl;
				if (atomic_updated == 0) atomic_updated = 1;
				// speedhack detection
				uint32_t unixts = std::strtoul(aes_decrypt(o1.get<jsonxx::String>("ts")).c_str(), NULL, 0);
				if (rem_unixts!=0) {
					uint32_t ts_diff = unixts - rem_unixts;
					if (ts_diff < 30 || ts_diff > 90) {
						LogTS(TEXT_BRED) << "[MALLOB] INVALID TIMESTAMP" << std::endl;
						invalid_timestamp_cnt++;
						if (invalid_timestamp_cnt>20) dynex_quit_flag = true;
					} else {
						invalid_timestamp_cnt = 0;
					}
				}
				rem_unixts = unixts;
			} else {
				errors++;
				LogTS(TEXT_BRED) << "[MALLOB] ATOMIC STATUS UPDATE: FAILED " << std::endl;
				rem_unixts = 0;
				if (errors > MAX_ATOMIC_ERR) dynex_quit_flag = true;
			}
		}
		auto t3 = std::chrono::steady_clock::now();
		int delay = std::chrono::duration_cast<std::chrono::milliseconds>(t3 - t2).count();
		if (60000 - delay > 0) std::this_thread::sleep_for(std::chrono::milliseconds(60000 - delay));
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
void gpu_runner(int device_id, int dev) {
	gpuErrchk(hipSetDevice(device_id));
	int num_jobs_gpu = num_jobs[dev];

	if (debug) LogTS(TEXT_BCYAN) << "[GPU " << device_id << "] STARTING ODE INTEGRATION..." << std::endl;
	if (debug) LogTS() << "[GPU " << device_id << "] PARAMETERS: α=" << dmm_alpha << " β=" << dmm_beta << " γ=" << dmm_gamma
			<< " ε=" << dmm_delta << " δ=" << dmm_epsilon << " ζ=" << dmm_zeta << " initial d_t=" << init_dt << std::endl;

	state_struct* h_state = (state_struct*)calloc(sizeof(state_struct) + n*sizeof(bool), 1);

	uint64_cu prev_steps = 0;
	bool gpu_solved = false;
	int global_loc = m;
	PRECISION global_energy = (PRECISION)m;
	auto t1 = std::chrono::steady_clock::now();
	auto t2 = t1;
	float hashrate = 0.0;
	bool terminal = false;
	bool newminima = false;
	bool newenergy = false;

	// integration loop:
	while (!dynex_quit_flag) {

		// reset dxdt:
		gpu_reset_dxdt <<< numBlocks[dev], threadsPerBlock[dev] >>> (dev, num_jobs[dev], d_jobs_2[dev]);
		// gpu step:
		gpu_step <<< numBlocks[dev], threadsPerBlock[dev] >>> (dev, num_jobs[dev], d_jobs_2[dev], d_state[dev]);
		gpuErrchk(hipMemcpy(h_state, d_state[dev], sizeof(state_struct), hipMemcpyDeviceToHost));

		// update globals:
		if (h_state->loc < global_loc) {
			global_loc = h_state->loc; // update global loc in case
			newminima = true;
			atomic_fetch_min(&overall_loc, global_loc);
		}
		if (h_state->energy < global_energy) {
			global_energy = h_state->energy; // update global energy in case
			newenergy = true;
			atomic_fetch_min(&overall_energy, global_energy);
		}
		// solution found?
		if (h_state->loc == 0) {
			gpu_solved = true;
			terminal = true;
			gpuErrchk(hipMemcpy(h_state, d_state[dev], sizeof(state_struct) + n*sizeof(bool), hipMemcpyDeviceToHost));
		}

		if (atomic_updated == 1) dynexservice.update(num_jobs_gpu);

		// console?
		auto t3 = std::chrono::steady_clock::now();
		float passedtime = std::chrono::duration_cast<std::chrono::milliseconds>(t3 - t2).count()/1000.0;
		if (passedtime > HASHRATE_INTERVAL) terminal = true;

		// show status in terminal:
		if (terminal) {
			t2 = t3;
			float uptime = std::chrono::duration_cast<std::chrono::milliseconds>(t3 - t1).count()/1000.0;
			hashrate = (h_state->steps - prev_steps)*num_jobs_gpu/passedtime;
			overall_hashrates[dev] = factor*hashrate;
			prev_steps = h_state->steps;
			std::string loc_flag = (newminima) ? "*" : " ";
			std::string energy_flag = (newenergy) ? "*" : " ";
			int hashrate_disp = (int)overall_hashrates[dev];
			LogTS() << "[GPU " << device_id << "] DYNEX CHIPS " << num_jobs_gpu << " | STEPS " << h_state->steps << " | LOCAL MINIMA " << global_loc << loc_flag
				<< " | LOWEST ENERGY " << std::fixed << std::setprecision(2) << global_energy << energy_flag
				<< " | POUW HASHRATE " << hashrate_disp << " | UPTIME " << uptime << "s " << std::endl;
			terminal = false;
			newminima = false;
			newenergy = false;
		}

		if (gpu_solved) break;

		// apply rhs / ode integration step:
		gpu_euler <<< numBlocks[dev], threadsPerBlock[dev] >>> (dev, num_jobs[dev], d_jobs_2[dev]);
	}

	// ---------------------------------------------------------------------------------------------------------------------
	// solution found?
	if (gpu_solved) {
		dynex_quit_flag = true; // quit all threads

		LogTS(TEXT_BYELLOW) << "[GPU " << device_id << "] SOLUTION FOUND!" << std::endl;
		// verify solution
		bool verify_sat = true;
		for (int j = 0; j < m; j++) {
			int lita = cls[j*MAX_LIT_SYSTEM+0]; bool a_pol = lita > 0 ? true : false;
			int litb = cls[j*MAX_LIT_SYSTEM+1]; bool b_pol = litb > 0 ? true : false;
			int litc = cls[j*MAX_LIT_SYSTEM+2]; bool c_pol = litc > 0 ? true : false;
			if (h_state->solution[abs(lita) - 1] != a_pol && h_state->solution[abs(litb) - 1] != b_pol && h_state->solution[abs(litc) - 1] != c_pol) {
				LogTS(TEXT_BRED) << "[ERROR] CLAUSE " << j << "[" << lita << " " << litb << " " << litc << "] HAS ASSIGNMENT "
					<< h_state->solution[abs(lita) - 1] << " " << h_state->solution[abs(litb) - 1] << " " << h_state->solution[abs(litc) - 1] << std::endl;
				verify_sat = false;
				break;
			}
		}
		if (!verify_sat) {
			LogTS(TEXT_BRED) << "[ERROR] SOLUTION NOT CERTIFIED" << std::endl;
		} else {
			LogTS(TEXT_BGREEN) << "[INFO] SOLUTION IS CERTIFIED" << std::endl;

			// output solution
			std::stringstream _solution;
			for (int i=0; i<n; i++) {
				_solution << (h_state->solution[i] ? i+1 : (i+1)*-1) << " ";
			}
			Log(TEXT_YELLOW) << "v " << _solution.str() << std::endl;

			//write solution to file:
			std::string solfile = JOB_FILENAME + ".solution.txt";
			FILE* fs = fopen(solfile.c_str(), "w");
			if (fs) {
				fprintf(fs, "%s\n", MINING_ADDRESS.c_str());
				for (int i=0; i<n; i++) {
					fprintf(fs, "%d, ", (h_state->solution[i] ? i+1 : (i+1)*-1));
				}
				fclose(fs);
				LogTS() << "[INFO] SOLUTION WRITTEN TO " << solfile << std::endl;
			}

			// submit solution to Dynex:
			if (!testing) {
				if (upload_file(solfile)) {
					LogTS(TEXT_BGREEN) << "[INFO] SOLUTION SUBMITTED TO DYNEX" << std::endl;
				}
			}

			if (!testing) {
				// atomic update:
				std::vector<std::string> p1;
				p1.push_back(MALLOB_NETWORK_ID);
				p1.push_back(std::to_string(2)); //status = solved
				p1.push_back(std::to_string(overall_loc));
				p1.push_back(std::to_string(overall_energy));
				p1.push_back(std::to_string(0));
				jsonxx::Object o1 = mallob_mpi_command("ato", p1, 60);
				if (o1.get<jsonxx::Boolean>("updated")) {
					LogTS(TEXT_SILVER) << "[MALLOB] ATOMIC STATUS UPDATED" << std::endl;
				} else {
					LogTS(TEXT_BRED) << "[MALLOB] ATOMIC STATUS NOT UPDATED " << std::endl;
				}
			}
		}
	}
	free(h_state);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
/// start dynexsolve
///////////////////////////////////////////////////////////////////////////////////////////////////////////
bool run_dynexsolve_2(int start_from_job, int maximum_jobs, int steps_per_batch, int device_id) {

	// initiate overall loc and energy:
	overall_loc = m;
	overall_energy = (PRECISION)m;
	for (int i=0; i < MAX_NUM_GPUS; i++) overall_hashrates[i] = 0;

	// configure threads and blocks:
	for (int i = 0; i < nDevices; i++) {
		numBlocks[i] = INTENSITY ? INTENSITY : 8192; 
		threadsPerBlock[i] = abs(num_jobs[i] / numBlocks[i]);
		if (numBlocks[i] < 1) numBlocks[i] = 1;
		if (threadsPerBlock[i] < 1) threadsPerBlock[i] = 1;
		if (debug) {
			LogTS() << "[DEBUG] GPU " << i << " threadsPerBlock = " << threadsPerBlock[i] << " numBlocks = " << numBlocks[i] << std::endl;
		}
	}

	int use_random = true;
	// init states for GPU:
	num_jobs_all = init_states_2(device_id, maximum_jobs, use_random);
	if (!num_jobs_all) return false;

	LogTS(TEXT_BCYAN) << "[INFO] STARTING ODE INTEGRATION..." << std::endl;

	// spawn a thread for each GPU:
	std::vector<std::thread> threads;
	for (int dev = 0; dev < nDevices; dev++) {
		if (use_multi_gpu) device_id = dev;
		// only not disabled gpus:
		if (std::find(disabled_gpus.begin(), disabled_gpus.end(), device_id) == disabled_gpus.end()) {
			threads.push_back(std::thread (gpu_runner, device_id, dev));
		}
	}

	if (!testing) {
		if (!dynexservice.start(threads.size(), STRATUM_URL, STRATUM_PORT, MINING_ADDRESS, STRATUM_PASSWORD, MALLOB_NETWORK_ID)) {
			LogTS(TEXT_BRED) << "[ERROR] CANNOT START DYNEX SERVICE" << std::endl;
			return false;
		}
	}

	// ---------------------------------------------------------------------------------------------------------------------
	// reporting runner:
	std::thread rep_th(gpu_reporting_runner, device_id, num_jobs_all);
	rep_th.detach();

	for (auto &thread: threads) {
		thread.join();
	}

	// ---------------------------------------------------------------------------------------------------------------------
	// stop dynexservice:
	dynexservice.stop();

	// ---------------------------------------------------------------------------------------------------------------------
	if (!testing) {
		// atomic update:
		std::vector<std::string> p1;
		p1.push_back(MALLOB_NETWORK_ID);
		p1.push_back(std::to_string(3)); //status = cancelled
		p1.push_back(std::to_string(overall_loc));
		p1.push_back(std::to_string(overall_energy));
		p1.push_back(std::to_string(0));
		jsonxx::Object o1 = mallob_mpi_command("ato", p1, 60);
		if (o1.get<jsonxx::Boolean>("updated")) {
			LogTS(TEXT_SILVER) << "[MALLOB] ATOMIC STATUS UPDATED" << std::endl;
		} else {
			LogTS(TEXT_BRED) << "[MALLOB] ATOMIC STATUS UPDATE: FAILED "<< std::endl;
		}
	}

	// ---------------------------------------------------------------------------------------------------------------------

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();
	return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
/// command line handler
///////////////////////////////////////////////////////////////////////////////////////////////////////////
char* getCmdOption(char** begin, char** end, const std::string& option)
{
	char** itr = std::find(begin, end, option);
	if (itr != end && ++itr != end)
	{
		return *itr;
	}
	return 0;
}

bool cmdOptionExists(char** begin, char** end, const std::string& option)
{
	return std::find(begin, end, option) != end;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
/// graceful exit handler
///////////////////////////////////////////////////////////////////////////////////////////////////////////
void signalHandler( int signum ) {
	LogTS() << " CTRL+C Interrupt signal (" << signum << ") received. Quitting gracefully..." << std::endl;

	// update mallob that we interruped:
	if (MALLOB_ACTIVE) {
		/// MALLOB: update_job_atomic -> let mallob know that we are working ++++++++++++++++++++++++++++++++++++++++++++++
		std::vector<std::string> p5;
		//network_id, atomic_status, steps_per_run, steps, hr, hr_adj
		p5.push_back(MALLOB_NETWORK_ID);
		p5.push_back(std::to_string(ATOMIC_STATUS_INTERRUPTED));
		p5.push_back(std::to_string(0));
		p5.push_back(std::to_string(0));
		p5.push_back(std::to_string(0));
		p5.push_back(std::to_string(0));
		jsonxx::Object o5 = mallob_mpi_command("update_atomic", p5, 60);
		if (o5.get<jsonxx::Boolean>("updated")) {
			LogTS(TEXT_SILVER) << "[INFO] MALLOB: ATOMIC JOB UPDATED" << std::endl;
		}
	/// +++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	}

	// stop miners:
	if (!SKIP) dynex_quit_flag = true; // stop signal to GPU job manager and CPU jobs
	if (!SKIP) dynexservice.dynex_hasher_quit_flag = true; // stop signal to Dynex hasher service
	if (!SKIP) LogTS(TEXT_SILVER) << "[INFO] FINISHING UP WORK ON GPU..." << std::endl;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Main
///////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	LogTS() << "[INFO] ---------------------------------------------------------" << std::endl;
	LogTS(TEXT_SILVER) << "[INFO] DynexSolve v" << VERSION << "(" << REVISION << ") | Meaningful Mining " << std::endl;
	LogTS() << "[INFO] ---------------------------------------------------------" << std::endl;

	// parse command line options:

	//help command?
	if (cmdOptionExists(argv, argv + argc, "-h"))
	{
		std::cout << "HELP" << std::endl;
		std::cout << "usage: dynexsolve -mining-address <WALLET ADDR> [options]" << std::endl;
		std::cout << std::endl;
		std::cout << "-mining-address <WALLET ADDR>    wallet address to receive the rewards" << std::endl;

		std::cout << "-stratum-url <HOST>              host of the stratum pool" << std::endl;
		std::cout << "-stratum-port <PORT>             port of the stratum pool" << std::endl;
		std::cout << "-stratum-paymentid <PAYMENT ID>  payment ID to add to wallet address" << std::endl;
		std::cout << "-stratum-password <PASSWORD>     stratum password (f.e. child@worker1)" << std::endl;
		std::cout << "-stratum-diff <DIFFICULTY>       stratum difficulty" << std::endl;

		std::cout << "-no-cpu                          run no Dynex chips on CPU" << std::endl;
		std::cout << "-no-gpu                          run no Dynex chips on GPU (WARNING: MINING NOT POSSIBLE)" << std::endl;
		std::cout << "-mallob-endpoint <IP>            set the endpoint for the Dynex Malleable Load Balancer" << std::endl;
		std::cout << "-devices                         show GPU devices on this system" << std::endl;
		std::cout << "-deviceid <GPU ID>               which GPU to use (default: 0 = first one) when using 1 GPU" << std::endl;
		std::cout << "-multi-gpu                       uses all GPUs in the system (default: off)" << std::endl;
		std::cout << "-disable-gpu <ID,ID,ID>          disable certain GPUs (check -devices for IDs) when using multi-gpu" << std::endl;
		std::cout << "-maximum-chips <JOBS>            set maximum number of parallel Dynex Chips to be run on GPU (default: INT_MAX)" << std::endl;
		std::cout << "-steps-per-batch <STEPS>         set number of steps per batch (default: 10000, min 10000)" << std::endl;
		std::cout << "-start-from-job <JOB_NUM>        set the starting job number (default: 0)" << std::endl;
		std::cout << "-cpu-chips <INT>                 set number of CPU Dynex-Chips to run (default: 4)" << std::endl;
		std::cout << "-alpha <DOUBLE>                  set alpha value of ODE" << std::endl;
		std::cout << "-beta <DOUBLE>                   set beta value of ODE" << std::endl;
		std::cout << "-gamma <DOUBLE>                  set gamma value of ODE" << std::endl;
		std::cout << "-delta <DOUBLE>                  set detla value of ODE" << std::endl;
		std::cout << "-epsilon <DOUBLE>                set epsilon value of ODE" << std::endl;
		std::cout << "-zeta <DOUBLE>                   set zeta value of ODE" << std::endl;
		std::cout << "-init_dt <DOUBLE>                set initial dt value of ODE" << std::endl;
		std::cout << "-stats <FILENAME>                save stats in json format to file" << std::endl;
		std::cout << "-adj <DOUBLE,DOUBLE,DOUBLE>      adjust used mem amount per GPU (default: " << ADJ_DEFAULT << ")" << std::endl;
		std::cout << "-debug                           enable debugging output" << std::endl;
		std::cout << "-test <INPUTFILE>                test Dynex Chips locally" << std::endl;
		std::cout << "-h                               show help" << std::endl;
		return EXIT_SUCCESS;
	}

	//query devices?
	if (cmdOptionExists(argv, argv + argc, "-devices")) {
		int devnum = query_devices(-1);
		return EXIT_SUCCESS;
	}

	//test?
	char* tf = getCmdOption(argv, argv + argc, "-test");
	if (tf) {
		testing = true;
		testing_file = tf;
		MINING_ADDRESS = "XwnV1b9sULyFvmW8NGQyndJGWkF9eE13XKobuGvHUS4QFRrKH7Ze8tRFM6kPeLjLHyfLWPoo7r8RJKyqpcGxZHk32f2avgT4t";
		LogTS(TEXT_BGREEN) << "[INFO] TESTING ACTIVATED: " << testing_file << std::endl;
	}

	char* in = getCmdOption(argv, argv + argc, "-intensity");
	if (in) {
		INTENSITY = atoi(in);
		if (INTENSITY < 0) INTENSITY = 0;
		LogTS() << "[INFO] INTENSITY SET TO " << INTENSITY << std::endl;
	}

	//stratum
	char* surl = getCmdOption(argv, argv + argc, "-stratum-url");
	if (surl) {
		STRATUM_URL = surl;
		stratum = true;
		LogTS() << "[INFO] STRATUM PROTOCOL ENABLED " << std::endl;
		LogTS() << "[INFO] STRATUM URL SET TO " << STRATUM_URL << std::endl;
	}

	char* sport = getCmdOption(argv, argv + argc, "-stratum-port");
	if (sport) {
		STRATUM_PORT = atoi(sport);
		LogTS() << "[INFO] STRATUM PORT SET TO " << STRATUM_PORT << std::endl;
	}
	if (stratum && STRATUM_PORT <= 0) {
		LogTS(TEXT_BRED) << "[ERROR] INVALID PORT" << std::endl;
		return EXIT_FAILURE;
	}

	char* spay = getCmdOption(argv, argv + argc, "-stratum-paymentid");
	if (spay) {
		STRATUM_PAYMENT_ID = spay;
		LogTS() << "[INFO] STRATUM PAYMENT ID SET TO " << STRATUM_PAYMENT_ID << std::endl;
	}

	char* spass = getCmdOption(argv, argv + argc, "-stratum-password");
	if (spass) {
		STRATUM_PASSWORD = spass;
		LogTS() << "[INFO] STRATUM PASSWORD SET TO " << STRATUM_PASSWORD << std::endl;
	}

	char* sdiff = getCmdOption(argv, argv + argc, "-stratum-diff");
	if (sdiff) {
		STRATUM_DIFF = atoi(sdiff);
		LogTS() << "[INFO] STRATUM DIFF SET TO " << STRATUM_DIFF << std::endl;
	}

	//mining-address
	char* ma = getCmdOption(argv, argv + argc, "-mining-address");
	if (ma) {
		MINING_ADDRESS = ma + (stratum ? (STRATUM_PAYMENT_ID != "" ? "." + STRATUM_PAYMENT_ID : "") + (STRATUM_DIFF != 0 ? "." + std::to_string(STRATUM_DIFF) : "") : "");
		LogTS() << "[INFO] MINING ADDRESS SET TO " << MINING_ADDRESS << std::endl;
	}

	if (MINING_ADDRESS=="") {
		LogTS(TEXT_BRED) << "[ERROR] WALLET ADDRESS NOT SPECIFIED" << std::endl;
		return EXIT_FAILURE;
	}

	//mallob endpoint?
	char* me = getCmdOption(argv, argv + argc, "-mallob-endpoint");
	if (me) {
		mallob_endpoint = me;
		LogTS() << "[INFO] OPTION mallob-endpoint SET TO " << mallob_endpoint << std::endl;
	}

	//debugger?
	bool dynex_debugger = false;
	if (cmdOptionExists(argv, argv + argc, "-debug")) {
		dynex_debugger = true;
		debug = dynex_debugger;
		LogTS() << "[INFO] OPTION debug ACTIVATED" << std::endl;
	}

	//- multi - gpu
	if (cmdOptionExists(argv, argv + argc, "-multi-gpu")) {
		use_multi_gpu = true;
		LogTS() << "[INFO] OPTION multi-gpu ACTIVATED" << std::endl;
	}

	//disable gpu?
	bool disable_gpu = false;
	if (cmdOptionExists(argv, argv + argc, "-no-gpu")) {
		disable_gpu = true;
		LogTS() << "[INFO] OPTION no-gpu ACTIVATED - "; Log(TEXT_BRED) << "ONLY SEARCHING FOR SOLUTION REWARD" << std::endl;
	}

	//disable certain?
	char* dgp = getCmdOption(argv, argv + argc, "-disable-gpu");
	if (dgp) {
		if (use_multi_gpu) {
			std::string disable_gpus = dgp;
			std::vector<std::string>disabled_gpus_str = split(disable_gpus,',');
			for (int i=0; i<disabled_gpus_str.size(); i++) disabled_gpus.push_back(atoi(disabled_gpus_str[i].c_str()));
			LogTS() << "[INFO] OPTION disable-gpu SET TO " << disable_gpus << std::endl;
		} else  {
			LogTS(TEXT_BRED) << "[ERROR] Option -disable-gpu cannot be used without option -multi-gpu" << std::endl;
			return EXIT_FAILURE;
		}
	}

	//alpha, beta, gamma, delta, epsilon, zeta:
	
	char* a = getCmdOption(argv, argv + argc, "-alpha");
	if (a) {
		dmm_alpha = atof(a);
		LogTS() << "[INFO] OPTION alpha SET TO " << dmm_alpha << std::endl;
	}
	char* b = getCmdOption(argv, argv + argc, "-beta");
	if (b) {
		dmm_beta = atof(b);
		LogTS() << "[INFO] OPTION beta SET TO " << dmm_beta << std::endl;
	}
	char* g = getCmdOption(argv, argv + argc, "-gamma");
	if (g) {
		dmm_gamma = atof(g);
		LogTS() << "[INFO] OPTION gamma SET TO " << dmm_gamma << std::endl;
	}
	char* d = getCmdOption(argv, argv + argc, "-delta");
	if (d) {
		dmm_delta = atof(d);
		LogTS() << "[INFO] OPTION delta SET TO " << dmm_delta << std::endl;
	}
	char* e = getCmdOption(argv, argv + argc, "-epsilon");
	if (e) {
		dmm_epsilon = atof(e);
		LogTS() << "[INFO] OPTION epsilon SET TO " << dmm_epsilon << std::endl;
	}
	char* z = getCmdOption(argv, argv + argc, "-zeta");
	if (z) {
		dmm_zeta = atof(z);
		LogTS() << "[INFO] OPTION zeta SET TO " << dmm_zeta << std::endl;
	}
	char* dt = getCmdOption(argv, argv + argc, "-init_dt");
	if (dt) {
		init_dt = atof(dt);
		LogTS() << "[INFO] OPTION init_dt SET TO " << init_dt << std::endl;
	}

	std::vector<std::string>adj_gpu;
	char* da = getCmdOption(argv, argv + argc, "-adj");
	if (da) {
		adj_gpu = split(da,',');
	}
	float adj_last = ADJ_DEFAULT;
	for (int i=0; i < MAX_NUM_GPUS; i++) {
		float adj = (i < adj_gpu.size()) ? atof(adj_gpu[i].c_str()) : adj_last;
		if (adj < 0.8) adj = 0.8;
		ADJ[i] = adj;
		adj_last = adj;
	}

	//cpu_chips?
	int cpu_chips = 0;
	char* rc = getCmdOption(argv, argv + argc, "-cpu-chips");
	if (rc) {
		cpu_chips = atoi(rc);
		if (cpu_chips < 0) cpu_chips = 0;
		if (cpu_chips > std::thread::hardware_concurrency()) cpu_chips = std::thread::hardware_concurrency();
		LogTS() << "[INFO] OPTION cpu-chips SET TO " << cpu_chips << std::endl;
	}

	//disable cpu?
	if (cmdOptionExists(argv, argv + argc, "-no-cpu")) {
		cpu_chips = 0;
		LogTS() << "[INFO] OPTION no-cpu ACTIVATED" << std::endl;
	}

	//start_from_job specified?
	int start_from_job = 0;
	char* sfj = getCmdOption(argv, argv + argc, "-start-from-job");
	if (sfj) {
		start_from_job = atoi(sfj);
		LogTS() << "[INFO] OPTION start-from-job SET TO " << start_from_job << std::endl;
	}

	//maximum_chips specified?
	int maximum_jobs = INT_MAX;
	char* mj = getCmdOption(argv, argv + argc, "-maximum-chips");
	if (mj) {
		maximum_jobs = atoi(mj);
		LogTS() << "[INFO] OPTION maximum-chips SET TO " << maximum_jobs << std::endl;
	}

	//maximum_jobs specified?
	int steps_per_batch = 10000;
	char* spb = getCmdOption(argv, argv + argc, "-steps-per-batch");
	if (spb) {
		steps_per_batch = atoi(spb);
		if (steps_per_batch < 10000) steps_per_batch = 10000;
		LogTS() << "[INFO] OPTION steps-per-batch SET TO " << steps_per_batch << std::endl;
	}

	//deviceid specified?
	int device_id = 0;
	char* did = getCmdOption(argv, argv + argc, "-deviceid");
	if (did) {
		device_id = atoi(did);
		LogTS() << "[INFO] OPTION deviceid SET TO " << device_id << std::endl;
		use_multi_gpu = false;
	}

	char* st = getCmdOption(argv, argv + argc, "-stats");
	if (st) {
		std::ofstream fout(st);
		if (fout.is_open()) {
			STATS = st;
			LogTS() << "[INFO] OPTION stats SET TO " << STATS << std::endl;
			fout << "{ \"ver\": \"" << VERSION << REVISION << "\", \"hr\": " << 0 << ", \"ac\": " << 0 << ", \"rj\": " << 0 << ", \"uptime\": " << 0 << " } " << std::endl;
			fout.close();
		} else {
			LogTS(TEXT_BRED) << "[ERROR] Unable to create stats file: " << STATS << std::endl;
		}
	}
	// ------------------------------------ end command line parameters --------------------------------------------------------------------

	// single or multi gpu?:
	if (!disable_gpu) {
		hipGetDeviceCount(&nDevices);
		if (!use_multi_gpu) {
			nDevices = 1;
		} else {
			// multi gpu:
			LogTS(TEXT_SILVER) << "[INFO] MULTI-GPU ENABLED" << std::endl;
			device_id = -1;
		}
		query_devices(device_id);
	}

	curl_global_init(CURL_GLOBAL_DEFAULT);

	/// MALLOB ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	if (MALLOB_NETWORK_ID == "") {
		std::random_device rd;
		std::mt19937_64 gen(rd());
		std::uniform_int_distribution<uint64_t> dis;
		std::stringstream sstream;
		for (int i = 0; i < 4; i++) {
			sstream << std::setw(16) << std::setfill('0') << std::hex << dis(gen);
		}
		MALLOB_NETWORK_ID = sstream.str();
		srand(dis(gen));
	}

	std::string SHA3HASH{}; 

	// Register as new worker:
	if (!testing) {
		std::vector<std::string> p1;
		p1.push_back(VERSION);
		p1.push_back(MALLOB_NETWORK_ID);
		p1.push_back(MINING_ADDRESS+(STRATUM_PASSWORD!="" ? ":"+STRATUM_PASSWORD : ""));
		jsonxx::Object o1 = mallob_mpi_command("reg", p1, 60);
		if (o1.get<jsonxx::Boolean>("registered")) {
			LogTS(TEXT_BGREEN) << "[MALLOB] REGISTER WORKER: SUCCESS" << std::endl;
			MALLOB_ACTIVE = true;
		} else {
			LogTS(TEXT_BRED) << "[MALLOB] REGISTER WORKER: FAILED" << std::endl;
			return EXIT_FAILURE;
		}
		int CHIPS_AVAILABLE, CHIPS_REQUIRED;
		double JOB_FEE, JOB_SOLUTION_REWARD;
		if (o1.has<jsonxx::String>("data")) {
			jsonxx::Object data;
			if (!data.parse(aes_decrypt(o1.get<jsonxx::String>("data")))) {
				LogTS(TEXT_BRED) << "[MALLOB] REGISTER WORKER: INVALID DATA" << std::endl;
				return EXIT_FAILURE;
			}
			//if (debug) LogTS() << data.json();
			JOB_ID = data.get<jsonxx::Number>("id");
			CHIPS_AVAILABLE = data.get<jsonxx::Number>("chips_available");
			CHIPS_REQUIRED = data.get<jsonxx::Number>("chips_required");
			JOB_FILENAME = data.get<jsonxx::String>("filename");
			JOB_FEE = data.get<jsonxx::Number>("fee");
			JOB_SOLUTION_REWARD = data.get<jsonxx::Number>("reward");
			dmm_alpha = data.get<jsonxx::Number>("P1");
			dmm_beta = data.get<jsonxx::Number>("P2");
			dmm_gamma = data.get<jsonxx::Number>("P3");
			dmm_delta = data.get<jsonxx::Number>("P4");
			dmm_epsilon = data.get<jsonxx::Number>("P5");
			dmm_zeta = data.get<jsonxx::Number>("P6");
			init_dt = data.get<jsonxx::Number>("P7");
			CNF_DOWNLOADURL = data.get<jsonxx::String>("downloadurl");
			CNF_SOLUTIONURL = data.get<jsonxx::String>("solutionurl");
			CNF_SOLUTIONUSER = data.get<jsonxx::String>("solutionuser");
			factor = data.get<jsonxx::Number>("factor");
			if (data.has<jsonxx::String>("network_id")) {
				MALLOB_NETWORK_ID = data.get<jsonxx::String>("network_id");
			}
			SHA3HASH = data.get<jsonxx::String>("sha3");
		} else {
			LogTS(TEXT_BRED) << "[MALLOB] REGISTER WORKER: INVALID DATA" << std::endl;
			return EXIT_FAILURE;
		}

		LogTS(TEXT_SILVER) << "[MALLOB] JOB RECEIVED        : " << JOB_ID << std::endl;
		LogTS(TEXT_SILVER) << "[MALLOB] CHIPS AVAILABLE     : " << CHIPS_AVAILABLE << "/" << CHIPS_REQUIRED << std::endl;
		LogTS(TEXT_SILVER) << "[MALLOB] JOB FILENAME        : " << JOB_FILENAME << std::endl;
		LogTS(TEXT_SILVER) << "[MALLOB] JOB FEE             : BLOCK REWARD + " << JOB_FEE <<  " DNX" << std::endl;
		LogTS(TEXT_SILVER) << "[MALLOB] JOB SOLUTION REWARD : " << JOB_SOLUTION_REWARD <<  " DNX" << std::endl;
		LogTS(TEXT_SILVER) << "[MALLOB] PARAMETERS: α=" << dmm_alpha << " β=" << dmm_beta << " γ=" << dmm_gamma << " ε=" << dmm_delta
			<< " δ=" << dmm_epsilon << " ζ=" << dmm_zeta << " initial d_t=" << init_dt << std::endl;

		// double check; chips also available?
		if (CHIPS_AVAILABLE <= 0) {
			LogTS(TEXT_BRED) << "[MALLOB] NO JOBS AVAILABLE" << std::endl;
			return EXIT_FAILURE;
		}
	}

	LogTS() << "[MALLOB] NETWORK ID " << MALLOB_NETWORK_ID << std::endl;

	// sanity check: mallob_network_id 64 bytes?
	if (MALLOB_NETWORK_ID.size() != 64) {
		LogTS(TEXT_BRED) << "[ERROR] NETWORK ID HAS THE WRONG SIZE. ABORT" << std::endl;
		return EXIT_FAILURE;
	}

	//convert(MALLOB_NETWORK_ID.c_str(), network_id, sizeof(network_id));

	// testing?
	if (testing) JOB_FILENAME = testing_file;

	// file existing?
	if (!file_exists(JOB_FILENAME)) {
		LogTS() << "[MALLOB] DOWNLOADING JOB: " << JOB_FILENAME << std::endl;
		if (!download_file(JOB_FILENAME)) return EXIT_FAILURE;
		LogTS(TEXT_BGREEN) << "[MALLOB] JOB SUCCESSFULLY DOWNLOADED" << std::endl;
	}

	/// +++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

	if (!load_cnf(JOB_FILENAME, SHA3HASH)) {
		std::remove(JOB_FILENAME.c_str()); // delete broken file
		return EXIT_FAILURE;
	}
	LogTS() << "[INFO] FORMULATION LOADED" << std::endl;

	// run CPU dynex chips
	bool dnxret = dynexchip.start(cpu_chips, JOB_FILENAME, std::ref(dynex_quit_flag), dmm_alpha, dmm_beta, dmm_gamma, dmm_delta, dmm_epsilon, dmm_zeta, init_dt, dynex_debugger, steps_per_batch);

	if (disable_gpu) {
		while (!dynex_quit_flag) {
			std::this_thread::sleep_for(std::chrono::milliseconds(100));
		}
		return EXIT_SUCCESS;
	}

	// run GPU dynex chips:
	if (!run_dynexsolve_2(start_from_job, maximum_jobs, steps_per_batch, device_id)) {
		LogTS(TEXT_BRED) << "[ERROR] EXIT WITH ERROR" << std::endl;
		return EXIT_FAILURE;
	}

	auto t2 = std::chrono::steady_clock::now();
	LogTS() << "[INFO] WALL TIME: "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t0).count()
		<< "ms" << std::endl;

	curl_global_cleanup();

	LogTS() << "GOOD BYE!" << std::endl;

	return EXIT_SUCCESS;
}
